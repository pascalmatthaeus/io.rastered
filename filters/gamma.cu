#include "hip/hip_runtime.h"

// +++ CONSTANTS +++

__device__ __constant__ int iConstants[3];
__device__ __constant__ float fConstants[2];

// +++ TEXTURES +++

texture<float, 2, hipReadModeElementType> tex;

// +++ HELPERS +++

__device__ float2 saturate(float2 f)
{
    if (f.x > 1.0f) f.x = 1.0f;
    if (f.y > 1.0f) f.y = 1.0f;
    if (f.x < 0.0f) f.x = 0.0f;
    if (f.y < 0.0f) f.y = 0.0f;
    return f;
}

__device__ float lerp(float p1, float p2, float i)
{
    return p2*i+p1*(1.0f-i);
}


// +++ KERNEL FUNCTIONS +++

extern "C"
{
    __global__ void gammaPow(float *out, int param)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int n = iConstants[2];
        int u = iConstants[0];
        int v = iConstants[1];
        float2 pxSize = make_float2(fConstants[0],fConstants[1]);
        float x = (float)(i%u)/u;
        float y = ((int)i/u)/(float)v;  // cut x indices
        float2 xy = make_float2(x,y);

        out[i] = powf(tex2D(tex,xy.x,xy.y),1.0f/((float)param/100.0f));

        if (out[i]>1.0f) out[i]=1.0f;
    }
}
