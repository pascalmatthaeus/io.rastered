#include "hip/hip_runtime.h"

// +++ CONSTANTS +++

__device__ __constant__ int iConstants[3];
__device__ __constant__ float fConstants[2];

// +++ TEXTURES +++

texture<float, 2, hipReadModeElementType> tex;

// +++ HELPERS +++

__device__ float2 saturate(float2 f)
{
    if (f.x > 1.0f) f.x = 1.0f;
    if (f.y > 1.0f) f.y = 1.0f;
    if (f.x < 0.0f) f.x = 0.0f;
    if (f.y < 0.0f) f.y = 0.0f;
    return f;
}

__device__ float lerp(float p1, float p2, float i)
{
    return p2*i+p1*(1.0f-i);
}

__device__ float blurBox(int radius, float2 xy, int axis)
{
    float2 pxSize = make_float2(fConstants[0],fConstants[1]);
    float out = tex2D(tex,xy.x,xy.y);
    float count = 1.0f;

    for (float i=1.5f;i<(float)radius;i+=2.0f)
    {
        out+=tex2D(tex,xy.x+pxSize.x*i*(1-axis),xy.y+pxSize.y*i*axis);
        out+=tex2D(tex,xy.x-pxSize.x*i*(1-axis),xy.y-pxSize.y*i*axis);
        count+=2.0f;
    }

    out/=count;
    return out;
}

// +++ KERNEL FUNCTIONS +++

extern "C"
{
    __global__ void blurH(float *out, int param)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int n = iConstants[2];
        int u = iConstants[0];
        int v = iConstants[1];
        float2 pxSize = make_float2(fConstants[0],fConstants[1]);
        float x = (float)(i%u)/u;
        float y = ((int)i/u)/(float)v;  // cut x indices
        float2 xy = make_float2(x,y);

        out[i] = blurBox(param,xy,0);

        if (out[i]>1.0f) out[i]=1.0f;
    }

    __global__ void blurV(float *out, int param)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int n = iConstants[2];
        int u = iConstants[0];
        int v = iConstants[1];
        float2 pxSize = make_float2(fConstants[0],fConstants[1]);
        float x = (float)(i%u)/u;
        float y = ((int)i/u)/(float)v;  // cut x indices
        float2 xy = make_float2(x,y);

        out[i] = blurBox(param,xy,1);

        if (out[i]>1.0f) out[i]=1.0f;
    }
}
